
#include "CudaRandom.cuh"

void* CreateCurandGenerator(void* streamId) throw(int)
{
	hiprandGenerator_t* gen = (hiprandGenerator_t*) malloc(sizeof(hiprandGenerator_t));
	hipStream_t* strId = (hipStream_t*) streamId;

	CURAND_SAFE_CALL(hiprandCreateGenerator(gen, HIPRAND_RNG_PSEUDO_DEFAULT));
	CURAND_SAFE_CALL(hiprandSetStream(*gen, *strId));

	return (void*) gen;
}

void DestroyCurandGenerator(void* generator) throw(int)
{
	hiprandGenerator_t* gen = (hiprandGenerator_t*) generator;

	CURAND_SAFE_CALL(hiprandDestroyGenerator(*gen));

	free(generator);
}

void CurandGenerateUniform(void* generator, float* devicePointer, int length) throw(int)
{
	hiprandGenerator_t* gen = (hiprandGenerator_t*) generator;

	CURAND_SAFE_CALL(hiprandGenerateUniform(*gen, devicePointer, length));
}