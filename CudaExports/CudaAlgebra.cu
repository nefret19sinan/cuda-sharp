
#include "CudaAlgebra.cuh"

extern "C"
{
	void* CreateCublasHandle(void* streamId) throw(int)
	{
		hipStream_t* strId = (hipStream_t*) streamId;
		hipblasHandle_t* chandle = (hipblasHandle_t*) malloc(sizeof(hipblasHandle_t));

		CUBLAS_SAFE_CALL(hipblasCreate(chandle));
		CUBLAS_SAFE_CALL(hipblasSetStream(*chandle, *strId));

		return (void*) chandle;
	}

	void DestroyCublasHandle(void* cublasHandle) throw(int)
	{
		hipblasHandle_t* chandle = (hipblasHandle_t*) cublasHandle;
	
		CUBLAS_SAFE_CALL(hipblasDestroy(*chandle));
	
		free(cublasHandle);
	}

	void CublasScalarMult(void* cublasHandle, float* devicePointer, int elemSpacing, int length, float* scalar) throw(int)
	{
		hipblasHandle_t* chandle = (hipblasHandle_t*) cublasHandle;

		CUBLAS_SAFE_CALL(hipblasSscal(*chandle, length, scalar, devicePointer, elemSpacing));
	}
}