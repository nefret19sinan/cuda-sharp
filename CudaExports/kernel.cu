
#include "kernel.cuh"

void* CreateCudaStream() throw(int)
{
	hipStream_t* streamId = (hipStream_t*) malloc(sizeof(hipStream_t));
	CUDA_SAFE_CALL(hipStreamCreate(streamId));

	return (void*) streamId;
}

void DestroyCudaStream(void* streamId) throw(int)
{
	hipStream_t* strId = (hipStream_t*) streamId;

	CUDA_SAFE_CALL(hipStreamDestroy(*strId));
	free(streamId);
}

void CudaStreamSync(void* streamId) throw(int)
{
	hipStream_t* strId = (hipStream_t*) streamId;

	CUDA_SAFE_CALL(hipStreamSynchronize(*strId));
}