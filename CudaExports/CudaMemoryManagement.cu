
#include "CudaMemoryManagement.cuh"


void* CudaHostMalloc(int sizeInBytes) throw(int)
{
	void* p;
	CUDA_SAFE_CALL(hipHostAlloc((void**) &p, sizeInBytes, hipHostMallocPortable));

	return p;
}

void* CudaDeviceMalloc(int sizeInBytes) throw(int)
{
	void* p;
	CUDA_SAFE_CALL(hipMalloc((void**) &p, sizeInBytes));

	return p;
}

void CudaHostFree(void* p) throw(int)
{
	CUDA_SAFE_CALL(hipHostFree(p));
}

void CudaDeviceFree(void* devPtr) throw(int)
{
	CUDA_SAFE_CALL(hipFree(devPtr));
}

void CudaSetVector(int n, int elemSize, void* hostPointer, int incX, void* devicePointer, 
	int incY, void* streamId) throw(int)
{
	hipStream_t* str = (hipStream_t*) streamId;
	
	CUBLAS_SAFE_CALL(hipblasSetVectorAsync(n, elemSize, hostPointer, incX, devicePointer, incY, *str));
}

void CudaGetVector(int n, int elemSize, void* devicePointer, int incX, void* hostPointer,
	int incY, void* streamId) throw(int)
{
	hipStream_t* str = (hipStream_t*) streamId;

	CUBLAS_SAFE_CALL(hipblasGetVectorAsync(n, elemSize, devicePointer, incX, hostPointer, incY, *str));
}